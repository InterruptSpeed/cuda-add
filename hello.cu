#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU!\n");
}

int main(void)
{
    helloFromGPU<<<1, 1>>>();
    hipDeviceReset();
    return 0;
}