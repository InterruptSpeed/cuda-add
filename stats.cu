#include <hip/hip_runtime.h>


#include <stdio.h>

int main(int argc, char *argv[])
{
    int iDev = 0;
    hipDeviceProp_t iProp;
    hipGetDeviceProperties(&iProp, iDev);
    printf("Device %d: %s\n", iDev, iProp.name);
    printf("Number of multiprocessors: %d\n", iProp.multiProcessorCount);
    printf("Total amount of constant memory: %4.2f KB\n",
           iProp.totalConstMem / 1024.0);
    printf("Total amount of shared memory per block: %4.2f KB\n",
           iProp.sharedMemPerBlock / 1024.0);
    printf("Total number of registers available per block: %d\n",
           iProp.regsPerBlock);
    printf("Warp size: %d\n", iProp.warpSize);
    printf("Maximum number of threads per block: %d\n", iProp.maxThreadsPerBlock);
    printf("Maximum number of threads per multiprocessor : %d\n",
           iProp.maxThreadsPerMultiProcessor);
}